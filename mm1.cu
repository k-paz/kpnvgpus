
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Funkcja kernela CUDA do mnożenia macierzy
__global__ void matrixMultiply(int *a, int *b, int *c, int width) {
    // Pobierz identyfikator bloku i wątku
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Zmienna do przechowywania wyniku mnożenia
    int sum = 0;

    // Mnożenie elementów i sumowanie wyniku
    for (int i = 0; i < width; ++i) {
        sum += a[row * width + i] * b[i * width + col];
    }

    // Zapisz wynik w macierzy wynikowej
    c[row * width + col] = sum;
}

int main() {
    // Rozmiar macierzy (kwadratowej w tym przypadku)
    int width = 1024;
    int size = width * width * sizeof(int);

    // Alokuje pamięć dla macierzy na CPU
    int *h_a, *h_b, *h_c;
    h_a = (int*)malloc(size);
    h_b = (int*)malloc(size);
    h_c = (int*)malloc(size);

    // Inicjalizacja macierzy
    for (int i = 0; i < width * width; ++i) {
        h_a[i] = rand() % 10;
        h_b[i] = rand() % 10;
    }

    // Alokuje pamięć dla macierzy na GPU
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Kopiuje dane z CPU do GPU
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Definiuje rozmiar bloku i siatkę wątków CUDA
    dim3 blockSize(16, 16);
    dim3 gridSize(width / blockSize.x, width / blockSize.y);

    // Wywołuje kernel CUDA do mnożenia macierzy
    matrixMultiply<<<gridSize, blockSize>>>(d_a, d_b, d_c, width);

    // Kopiuje wynik z GPU do CPU
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Sprawdza wynik (opcjonalnie)
    // ... (dodaj kod sprawdzający)

    // Zwolnij pamięć
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

